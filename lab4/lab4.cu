
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8
#define X_BOUND 2
#define Y_BOUND 2

__constant__ char mask[MASK_N][MASK_X][MASK_Y] = { 
	{{ -1, -4, -6, -4, -1},
	{ -2, -8,-12, -8, -2},
	{  0,  0,  0,  0,  0}, 
	{  2,  8, 12,  8,  2}, 
	{  1,  4,  6,  4,  1}},
	{{ -1, -2,  0,  2,  1}, 
	{ -4, -8,  0,  8,  4}, 
	{ -6,-12,  0, 12,  6}, 
	{ -4, -8,  0,  8,  4}, 
	{ -1, -2,  0,  2,  1}} 
};

int read_png(const char* filename, unsigned char** image, unsigned* height, unsigned* width, unsigned* channels) {
	unsigned char sig[8];
	FILE* infile;
	infile = fopen(filename, "rb");

	fread(sig, 1, 8, infile);
	if (!png_check_sig(sig, 8))
			return 1;   /* bad signature */

	png_structp png_ptr;
	png_infop info_ptr;

	png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
	if (!png_ptr)
		return 4;   /* out of memory */

	info_ptr = png_create_info_struct(png_ptr);
	if (!info_ptr) {
		png_destroy_read_struct(&png_ptr, NULL, NULL);
		return 4;   /* out of memory */
	}

	png_init_io(png_ptr, infile);
	png_set_sig_bytes(png_ptr, 8);
	png_read_info(png_ptr, info_ptr);
	int bit_depth, color_type;
	png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

	png_uint_32  i, rowbytes;
	png_bytep  row_pointers[*height];
	png_read_update_info(png_ptr, info_ptr);
	rowbytes = png_get_rowbytes(png_ptr, info_ptr);
	*channels = (int)png_get_channels(png_ptr, info_ptr);

	if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
		png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
		return 3;
	}

	for (i = 0; i < *height; ++i)
		row_pointers[i] = *image + i * rowbytes;
	png_read_image(png_ptr, row_pointers);
	png_read_end(png_ptr, NULL);
	return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, const unsigned channels) {
	FILE* fp = fopen(filename, "wb");
	png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
	png_infop info_ptr = png_create_info_struct(png_ptr);
	png_init_io(png_ptr, fp);
	png_set_IHDR(png_ptr, info_ptr, width, height, 8,
							PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
							PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
	png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
	png_write_info(png_ptr, info_ptr);
	png_set_compression_level(png_ptr, 1);

	png_bytep row_ptr[height];
	for (int i = 0; i < height; ++ i) {
		row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
	}
	png_write_image(png_ptr, row_ptr);
	png_write_end(png_ptr, NULL);
	png_destroy_write_struct(&png_ptr, &info_ptr);
	fclose(fp);
}

__global__ void sobel(unsigned char* s, unsigned char* t, unsigned height, unsigned width, unsigned channels) {
	int val[MASK_N][3];

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= width) return;

	__shared__ unsigned char sR[5][260];
	__shared__ unsigned char sG[5][260];
	__shared__ unsigned char sB[5][260];

	for (int y = 0; y < height; ++y) {
		if (y == 0) {
			for (int v = -Y_BOUND; v <= Y_BOUND; ++v) {
				if (y + v >= 0 && y + v < height) {
					int idx = channels * (width * (y + v) + x);
					sR[v + Y_BOUND][threadIdx.x + X_BOUND] = s[idx + 2];
					sG[v + Y_BOUND][threadIdx.x + X_BOUND] = s[idx + 1];
					sB[v + Y_BOUND][threadIdx.x + X_BOUND] = s[idx];

					if (threadIdx.x == 0) {
						if (x - 2 >= 0) {
							int idx1 = channels * (width * (y + v) + (x - 2));
							sR[v + Y_BOUND][0] = s[idx1 + 2];
							sG[v + Y_BOUND][0] = s[idx1 + 1];
							sB[v + Y_BOUND][0] = s[idx1];
						}
						if (x - 1 >= 0) {
							int idx1 = channels * (width * (y + v) + (x - 1));
							sR[v + Y_BOUND][1] = s[idx1 + 2];
							sG[v + Y_BOUND][1] = s[idx1 + 1];
							sB[v + Y_BOUND][1] = s[idx1];
						}
					}

					if (threadIdx.x == blockDim.x - 1) {
						if (x + 1 < width) {
							int idx1 = channels * (width * (y + v) + (x + 1));
							sR[v + Y_BOUND][threadIdx.x + X_BOUND + 1] = s[idx1 + 2];
							sG[v + Y_BOUND][threadIdx.x + X_BOUND + 1] = s[idx1 + 1];
							sB[v + Y_BOUND][threadIdx.x + X_BOUND + 1] = s[idx1];
						}
						if (x + 2 < width) {
							int idx1 = channels * (width * (y + v) + (x + 2));
							sR[v + Y_BOUND][threadIdx.x + X_BOUND + 2] = s[idx1 + 2];
							sG[v + Y_BOUND][threadIdx.x + X_BOUND + 2] = s[idx1 + 1];
							sB[v + Y_BOUND][threadIdx.x + X_BOUND + 2] = s[idx1];
						}
					}
				}
			}
		} else {
			for (int i = 0; i < 4; i++) {
				sR[i][threadIdx.x + X_BOUND] = sR[i + 1][threadIdx.x + X_BOUND];
				sG[i][threadIdx.x + X_BOUND] = sG[i + 1][threadIdx.x + X_BOUND];
				sB[i][threadIdx.x + X_BOUND] = sB[i + 1][threadIdx.x + X_BOUND];
				if (threadIdx.x == 0) {
					sR[i][0] = sR[i + 1][0];
					sG[i][0] = sG[i + 1][0];
					sB[i][0] = sB[i + 1][0];
					sR[i][1] = sR[i + 1][1];
					sG[i][1] = sG[i + 1][1];
					sB[i][1] = sB[i + 1][1];
				}
				if (threadIdx.x == blockDim.x - 1) {
					sR[i][threadIdx.x + X_BOUND + 1] = sR[i + 1][threadIdx.x + X_BOUND + 1];
					sG[i][threadIdx.x + X_BOUND + 1] = sG[i + 1][threadIdx.x + X_BOUND + 1];
					sB[i][threadIdx.x + X_BOUND + 1] = sB[i + 1][threadIdx.x + X_BOUND + 1];
					sR[i][threadIdx.x + X_BOUND + 2] = sR[i + 1][threadIdx.x + X_BOUND + 2];
					sG[i][threadIdx.x + X_BOUND + 2] = sG[i + 1][threadIdx.x + X_BOUND + 2];
					sB[i][threadIdx.x + X_BOUND + 2] = sB[i + 1][threadIdx.x + X_BOUND + 2];
				}
			}

			int v = Y_BOUND;
			if (y + v >= 0 && y + v < height) {
				int idx = channels * (width * (y + v) + x);
				sR[v + Y_BOUND][threadIdx.x + X_BOUND] = s[idx + 2];
				sG[v + Y_BOUND][threadIdx.x + X_BOUND] = s[idx + 1];
				sB[v + Y_BOUND][threadIdx.x + X_BOUND] = s[idx];

				if (threadIdx.x == 0) {
					if (x - 2 >= 0) {
						int idx1 = channels * (width * (y + v) + (x - 2));
						sR[v + Y_BOUND][0] = s[idx1 + 2];
						sG[v + Y_BOUND][0] = s[idx1 + 1];
						sB[v + Y_BOUND][0] = s[idx1];
					}
					if (x - 1 >= 0) {
						int idx1 = channels * (width * (y + v) + (x - 1));
						sR[v + Y_BOUND][1] = s[idx1 + 2];
						sG[v + Y_BOUND][1] = s[idx1 + 1];
						sB[v + Y_BOUND][1] = s[idx1];
					}
				}

				if (threadIdx.x == blockDim.x - 1) {
					if (x + 1 < width) {
						int idx1 = channels * (width * (y + v) + (x + 1));
						sR[v + Y_BOUND][threadIdx.x + X_BOUND + 1] = s[idx1 + 2];
						sG[v + Y_BOUND][threadIdx.x + X_BOUND + 1] = s[idx1 + 1];
						sB[v + Y_BOUND][threadIdx.x + X_BOUND + 1] = s[idx1];
					}
					if (x + 2 < width) {
						int idx1 = channels * (width * (y + v) + (x + 2));
						sR[v + Y_BOUND][threadIdx.x + X_BOUND + 2] = s[idx1 + 2];
						sG[v + Y_BOUND][threadIdx.x + X_BOUND + 2] = s[idx1 + 1];
						sB[v + Y_BOUND][threadIdx.x + X_BOUND + 2] = s[idx1];
					}
				}
			}
		}

		__syncthreads();

		for (int i = 0; i < MASK_N; ++i) {
			val[i][2] = 0;
			val[i][1] = 0;
			val[i][0] = 0;

			for (int v = -Y_BOUND; v <= Y_BOUND; ++v) {
				for (int u = -X_BOUND; u <= X_BOUND; ++u) {
					if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
						int idx = threadIdx.x + u + X_BOUND;
						const unsigned char R = sR[v + Y_BOUND][idx];
						const unsigned char G = sG[v + Y_BOUND][idx];
						const unsigned char B = sB[v + Y_BOUND][idx];

						val[i][2] += R * mask[i][u + X_BOUND][v + Y_BOUND];
						val[i][1] += G * mask[i][u + X_BOUND][v + Y_BOUND];
						val[i][0] += B * mask[i][u + X_BOUND][v + Y_BOUND];
					}
				}
			}
		}

		float totalR = 0;
		float totalG = 0;
		float totalB = 0;
		for (int i = 0; i < MASK_N; ++i) {
			totalR += val[i][2] * val[i][2];
			totalG += val[i][1] * val[i][1];
			totalB += val[i][0] * val[i][0];
		}

		totalR = sqrt(totalR) / SCALE;
		totalG = sqrt(totalG) / SCALE;
		totalB = sqrt(totalB) / SCALE;
		const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
		const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
		const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
		t[channels * (width * y + x) + 2] = cR;
		t[channels * (width * y + x) + 1] = cG;
		t[channels * (width * y + x) + 0] = cB;

		__syncthreads();
	}
}

int main(int argc, char** argv) {
	assert(argc == 3);
	unsigned height, width, channels;
	unsigned char* host_s = NULL;
	read_png(argv[1], &host_s, &height, &width, &channels);
	unsigned char* host_t = (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));

	unsigned char* device_s;
	unsigned char* device_t;
	hipMalloc((void**)&device_s, height * width * channels * sizeof(unsigned char));
	hipMalloc((void**)&device_t, height * width * channels * sizeof(unsigned char));

	hipMemcpy(device_s, host_s, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

	const int threads_per_block = 256;
	const int number_of_blocks = (width + threads_per_block - 1) / threads_per_block;
	sobel<<<number_of_blocks, threads_per_block>>>(device_s, device_t, height, width, channels);

	hipMemcpy(host_t, device_t, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

	write_png(argv[2], host_t, height, width, channels);

	return 0;
}

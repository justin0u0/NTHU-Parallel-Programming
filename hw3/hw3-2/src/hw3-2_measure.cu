// Author: justin0u0<mail@justin0u0.com>
//
// Finished basic blocked floyd warshall.
// Optimization:
//  - Calculate 4 position in a single thread to fully utilize shared memory.
//  - Unroll the loop
//  - Use cudaMallocPitch + cudaMemcpy2D
//  - Do `fread` only once to reduce times of system call


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

void startSpan(std::chrono::steady_clock::time_point& t) {
	t = std::chrono::steady_clock::now();
}

void endSpan(std::chrono::steady_clock::time_point& t, int& total) {
	total += std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count();
}

const int INF = ((1 << 30) - 1);
const int BLOCK_SIZE = 64;
const int HBS = 32; // define the half block size

void handleInput(const char* inputFile, int& n, int& m, int& origN, int** hostD) {
	FILE* file = fopen(inputFile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

	origN = n;
	if (n % BLOCK_SIZE != 0) {
		n = n + (BLOCK_SIZE - n % BLOCK_SIZE);
	}

	*hostD = (int*)malloc(n * n * sizeof(int));

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			(*hostD)[i * n + j] = (i == j) ? 0 : INF;
		}
	}

	int* edges = (int*)malloc(3 * m * sizeof(int));
	fread(edges, sizeof(int), 3 * m, file);
	for (int i = 0; i < m; ++i) {
		(*hostD)[edges[i * 3] * n + edges[i * 3 + 1]] = edges[i * 3 + 2];
	}

	// free(edges);

	fclose(file);
}

void handleOutput(const char* outputFile, const int n, const int origN, int* hostD) {
	FILE* file = fopen(outputFile, "w");

	for (int i = 0; i < origN; ++i) {
		fwrite(hostD + i * n, sizeof(int), origN, file);
	}
	// fwrite(hostD, sizeof(int), origN * origN, file);

	fclose(file);
}

__global__ void blockedFloydWarshallPhase1(int n, int blockId, int* d, size_t pitch) {
	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	const unsigned int &x = threadIdx.x;
	const unsigned int &y = threadIdx.y;

	// load the block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	cacheD[y][x] = d[i * pitch + j];
	cacheD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];
	__syncthreads();

	// compute phase 1 - dependent phase
	#pragma unroll 32
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		// using cuda min
		cacheD[y][x] = min(cacheD[y][x], cacheD[y][k] + cacheD[k][x]);
		cacheD[y + HBS][x] = min(cacheD[y + HBS][x], cacheD[y + HBS][k] + cacheD[k][x]);
		cacheD[y][x + HBS] = min(cacheD[y][x + HBS], cacheD[y][k] + cacheD[k][x + HBS]);
		cacheD[y + HBS][x + HBS] = min(cacheD[y + HBS][x + HBS], cacheD[y + HBS][k] + cacheD[k][x + HBS]);

		// using branch
		/*
		cacheD[y][x] = (cacheD[y][x] < cacheD[y][k] + cacheD[k][x]) ? cacheD[y][x] : cacheD[y][k] + cacheD[k][x];
		cacheD[y + HBS][x] = (cacheD[y + HBS][x] < cacheD[y + HBS][k] + cacheD[k][x]) ? cacheD[y + HBS][x] : cacheD[y + HBS][k] + cacheD[k][x];
		cacheD[y][x + HBS] = (cacheD[y][x + HBS] < cacheD[y][k] + cacheD[k][x + HBS]) ? cacheD[y][x + HBS] : cacheD[y][k] + cacheD[k][x + HBS];
		cacheD[y + HBS][x + HBS] = (cacheD[y + HBS][x + HBS] < cacheD[y + HBS][k] + cacheD[k][x + HBS]) ? cacheD[y + HBS][x + HBS] : cacheD[y + HBS][k] + cacheD[k][x + HBS];
		*/

		__syncthreads();
	}

	// load shared memory back to the global memory
	d[i * pitch + j] = cacheD[y][x];
	d[(i + HBS) * pitch + j] = cacheD[y + HBS][x];
	d[i * pitch + (j + HBS)] = cacheD[y][x + HBS];
	d[(i + HBS) * pitch + (j + HBS)] = cacheD[y + HBS][x + HBS];
}

__global__ void blockedFloydWarshallPhase2(int n, int blockId, int* d, size_t pitch) {
	// skipping the base block (from phase 1)
	if (blockIdx.x == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	const unsigned int& x = threadIdx.x;
	const unsigned int& y = threadIdx.y;

	// load the base block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;

	__shared__ int cacheBaseD[BLOCK_SIZE][BLOCK_SIZE];
	cacheBaseD[y][x] = d[i * pitch + j];
	cacheBaseD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheBaseD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheBaseD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	// load the target block of same row into shared memory
	i = y + blockId * BLOCK_SIZE;
	j = x + blockIdx.x * BLOCK_SIZE;

	__shared__ int cacheRowD[BLOCK_SIZE][BLOCK_SIZE];
	cacheRowD[y][x] = d[i * pitch + j];
	cacheRowD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheRowD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheRowD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	// load the target block of same column into shared memory
	i = y + blockIdx.x * BLOCK_SIZE;
	j = x + blockId * BLOCK_SIZE;

	__shared__ int cacheColD[BLOCK_SIZE][BLOCK_SIZE];
	cacheColD[y][x] = d[i * pitch + j];
	cacheColD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheColD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheColD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	__syncthreads();

	// compute phase 2 - partial dependent phase
	#pragma unroll 32
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		// using cuda min
		cacheRowD[y][x] = min(cacheRowD[y][x], cacheBaseD[y][k] + cacheRowD[k][x]);
		cacheRowD[y + HBS][x] = min(cacheRowD[y + HBS][x], cacheBaseD[y + HBS][k] + cacheRowD[k][x]);
		cacheRowD[y][x + HBS] = min(cacheRowD[y][x + HBS], cacheBaseD[y][k] + cacheRowD[k][x + HBS]);
		cacheRowD[y + HBS][x + HBS] = min(cacheRowD[y + HBS][x + HBS], cacheBaseD[y + HBS][k] + cacheRowD[k][x + HBS]);

		cacheColD[y][x] = min(cacheColD[y][x], cacheColD[y][k] + cacheBaseD[k][x]);
		cacheColD[y + HBS][x] = min(cacheColD[y + HBS][x], cacheColD[y + HBS][k] + cacheBaseD[k][x]);
		cacheColD[y][x + HBS] = min(cacheColD[y][x + HBS], cacheColD[y][k] + cacheBaseD[k][x + HBS]);
		cacheColD[y + HBS][x + HBS] = min(cacheColD[y + HBS][x + HBS], cacheColD[y + HBS][k] + cacheBaseD[k][x + HBS]);

		// using branch
		/*
		cacheRowD[y][x] = (cacheRowD[y][x] < cacheBaseD[y][k] + cacheRowD[k][x]) ? cacheRowD[y][x] : cacheBaseD[y][k] + cacheRowD[k][x];
		cacheRowD[y + HBS][x] = (cacheRowD[y + HBS][x] < cacheBaseD[y + HBS][k] + cacheRowD[k][x]) ? cacheRowD[y + HBS][x] : cacheBaseD[y + HBS][k] + cacheRowD[k][x];
		cacheRowD[y][x + HBS] = (cacheRowD[y][x + HBS] < cacheBaseD[y][k] + cacheRowD[k][x + HBS]) ? cacheRowD[y][x + HBS] : cacheBaseD[y][k] + cacheRowD[k][x + HBS];
		cacheRowD[y + HBS][x + HBS] = (cacheRowD[y + HBS][x + HBS] < cacheBaseD[y + HBS][k] + cacheRowD[k][x + HBS]) ? cacheRowD[y + HBS][x + HBS] : cacheBaseD[y + HBS][k] + cacheRowD[k][x + HBS];

		cacheColD[y][x] = (cacheColD[y][x] < cacheColD[y][k] + cacheBaseD[k][x]) ? cacheColD[y][x] : cacheColD[y][k] + cacheBaseD[k][x];
		cacheColD[y + HBS][x] = (cacheColD[y + HBS][x] < cacheColD[y + HBS][k] + cacheBaseD[k][x]) ? cacheColD[y + HBS][x] : cacheColD[y + HBS][k] + cacheBaseD[k][x];
		cacheColD[y][x + HBS] = (cacheColD[y][x + HBS] < cacheColD[y][k] + cacheBaseD[k][x + HBS]) ? cacheColD[y][x + HBS] : cacheColD[y][k] + cacheBaseD[k][x + HBS];
		cacheColD[y + HBS][x + HBS] = (cacheColD[y + HBS][x + HBS] < cacheColD[y + HBS][k] + cacheBaseD[k][x + HBS]) ? cacheColD[y + HBS][x + HBS] : cacheColD[y + HBS][k] + cacheBaseD[k][x + HBS];
		*/
	}

	// load shared memory back to the global memory

	// i = y + blockIdx.x * BLOCK_SIZE;
	// j = x + blockId * BLOCK_SIZE;
	d[i * pitch + j] = cacheColD[y][x];
	d[(i + HBS) * pitch + j] = cacheColD[y + HBS][x];
	d[i * pitch + (j + HBS)] = cacheColD[y][x + HBS];
	d[(i + HBS) * pitch + (j + HBS)] = cacheColD[y + HBS][x + HBS];

	i = y + blockId * BLOCK_SIZE;
	j = x + blockIdx.x * BLOCK_SIZE;
	d[i * pitch + j] = cacheRowD[y][x];
	d[(i + HBS) * pitch + j] = cacheRowD[y + HBS][x];
	d[i * pitch + (j + HBS)] = cacheRowD[y][x + HBS];
	d[(i + HBS) * pitch + (j + HBS)] = cacheRowD[y + HBS][x + HBS];
}

__global__ void blockedFloydWarshallPhase3(int n, int blockId, int* d, size_t pitch) {
	// skipping the base blocks (from phase 1, 2)
	if (blockIdx.x == blockId || blockIdx.y == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	const unsigned int& x = threadIdx.x;
	const unsigned int& y = threadIdx.y;

	int i; // y + blockIdx.y * BLOCK_SIZE;
	int j; // x + blockIdx.x * BLOCK_SIZE;

	// load the base column block (same row) into shared memory
	__shared__ int cacheBaseColD[BLOCK_SIZE][BLOCK_SIZE];
	i = y + blockIdx.y * BLOCK_SIZE;
	j = x + blockId * BLOCK_SIZE;
	cacheBaseColD[y][x] = d[i * pitch + j];
	cacheBaseColD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheBaseColD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheBaseColD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	// load the base row block (same column) into shared memory
	__shared__ int cacheBaseRowD[BLOCK_SIZE][BLOCK_SIZE];
	i = y + blockId * BLOCK_SIZE;
	j = x + blockIdx.x * BLOCK_SIZE;
	cacheBaseRowD[y][x] = d[i * pitch + j];
	cacheBaseRowD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheBaseRowD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheBaseRowD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];
	__syncthreads();

	// load the target block into shared memory
	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	i = y + blockIdx.y * BLOCK_SIZE;
	j = x + blockIdx.x * BLOCK_SIZE;
	cacheD[y][x] = d[i * pitch + j];
	cacheD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	// compute phase 3 - independence phase
	#pragma unroll 32
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		// using cuda min
		cacheD[y][x] = min(cacheD[y][x], cacheBaseColD[y][k] + cacheBaseRowD[k][x]);
		cacheD[y + HBS][x] = min(cacheD[y + HBS][x], cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x]);
		cacheD[y][x + HBS] = min(cacheD[y][x + HBS], cacheBaseColD[y][k] + cacheBaseRowD[k][x + HBS]);
		cacheD[y + HBS][x + HBS] = min(cacheD[y + HBS][x + HBS], cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x + HBS]);

		// using branch
		/*
		cacheD[y][x] = (cacheD[y][x] < cacheBaseColD[y][k] + cacheBaseRowD[k][x]) ? cacheD[y][x] : cacheBaseColD[y][k] + cacheBaseRowD[k][x];
		cacheD[y + HBS][x] = (cacheD[y + HBS][x] < cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x]) ? cacheD[y + HBS][x] : cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x];
		cacheD[y][x + HBS] = (cacheD[y][x + HBS] < cacheBaseColD[y][k] + cacheBaseRowD[k][x + HBS]) ? cacheD[y][x + HBS] : cacheBaseColD[y][k] + cacheBaseRowD[k][x + HBS];
		cacheD[y + HBS][x + HBS] = (cacheD[y + HBS][x + HBS] < cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x + HBS]) ? cacheD[y + HBS][x + HBS] : cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x + HBS];
		*/
	}

	// load shared memory back to the global memory
	d[i * pitch + j] = cacheD[y][x];
	d[(i + HBS) * pitch + j] = cacheD[y + HBS][x];
	d[i * pitch + (j + HBS)] = cacheD[y][x + HBS];
	d[(i + HBS) * pitch + (j + HBS)] = cacheD[y + HBS][x + HBS];
}

int main(int argc, char** argv) {
	std::chrono::steady_clock::time_point totalT, computeT;
	int totalTime = 0, computeTime = 0;;
	startSpan(totalT);

	int n, m, origN;

	int* hostD;
	handleInput(argv[1], n, m, origN, &hostD);

	// pinned the host memory to accerlate cudaMemcpy
	hipHostRegister(hostD, n * n * sizeof(int), hipHostMallocDefault);

	int* deviceD;
	/* zero copy */
	// cudaHostGetDevicePointer((void**)&deviceD, (void*)hostD, cudaHostRegisterDefault);

	/* normal cuda malloc + memcpy */
	// cudaMalloc((void**)&deviceD, n * n * sizeof(int));
	// cudaMemcpy(deviceD, hostD, n * n * sizeof(int), cudaMemcpyHostToDevice);

	/* cudaMallocPitch + cudaMemcpy2D */
	size_t pitch;
	hipMallocPitch((void**)&deviceD, &pitch, n * sizeof(int), n);
	hipMemcpy2D(deviceD, pitch, hostD, n * sizeof(int), n * sizeof(int), n, hipMemcpyHostToDevice);

	/* blocked floyd warshall */

	// number of blocks is numberOfBlocks * numberOfBlocks
	// int numberOfBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	int numberOfBlocks = n / BLOCK_SIZE;

	dim3 gridPhase1(1, 1);
	dim3 gridPhase2(numberOfBlocks, 1);
	dim3 gridPhase3(numberOfBlocks, numberOfBlocks);
	dim3 threadsPerBlock(HBS, HBS);

	startSpan(computeT);
	for (int blockId = 0; blockId < numberOfBlocks; ++blockId) {
		blockedFloydWarshallPhase1<<<gridPhase1, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase2<<<gridPhase2, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase3<<<gridPhase3, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
	}
	hipDeviceSynchronize();
	endSpan(computeT, computeTime);

	/* zero copy */

	/* normal cuda memcpy */
	// cudaMemcpy(hostD, deviceD, n * n * sizeof(int), cudaMemcpyDeviceToHost);

	/* cudaMemcpy2D */
	hipMemcpy2D(hostD, n * sizeof(int), deviceD, pitch, n * sizeof(int), n, hipMemcpyDeviceToHost);
	hipFree(deviceD);

	handleOutput(argv[2], n, origN, hostD);

	// this slow down the execution ...
	// free(hostD);

	endSpan(totalT, totalTime);

	printf("total time: %d (ms)\n", totalTime);
	printf("compute time: %d (ms)\n", computeTime);
	return 0;
}

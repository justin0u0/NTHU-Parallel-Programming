// Author: justin0u0<mail@justin0u0.com>
//
// Finished basic blocked floyd warshall.
// Optimization:
//   - Calculate 4 position in a single thread to fully utilize shared memory.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = ((1 << 30) - 1);
const int BLOCK_SIZE = 64;
const int HBS = 32; // define the half block size

void handleInput(const char* inputFile, int& n, int& m, int& origN, int** hostD) {
	FILE* file = fopen(inputFile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

	origN = n;
	if (n % BLOCK_SIZE != 0) {
		n = n + (BLOCK_SIZE - n % BLOCK_SIZE);
	}

	*hostD = (int*)malloc(n * n * sizeof(int));

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			(*hostD)[i * n + j] = (i == j) ? 0 : INF;
		}
	}

	int edge[3];
	for (int i = 0; i < m; ++i) {
		fread(edge, sizeof(int), 3, file);
		(*hostD)[edge[0] * n + edge[1]] = edge[2];
	}
	fclose(file);
}

void handleOutput(const char* outputFile, const int n, const int origN, int* hostD) {
	FILE* file = fopen(outputFile, "w");
	for (int i = 0; i < origN; ++i) {
		fwrite(hostD + i * n, sizeof(int), origN, file);
	}
	// fwrite(hostD, sizeof(int), n * n, file);
	fclose(file);
}

__global__ void blockedFloydWarshallPhase1(int n, int blockId, int* d, size_t pitch) {
	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// load the block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	cacheD[y][x] = d[i * pitch + j];
	cacheD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];
	__syncthreads();

	// compute phase 1 - dependent phase
	#pragma unroll 32
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		// cacheD[y][x] = min(cacheD[y][x], cacheD[y][k] + cacheD[k][x]);
		cacheD[y][x] = (cacheD[y][x] < cacheD[y][k] + cacheD[k][x]) ? cacheD[y][x] : cacheD[y][k] + cacheD[k][x];
		cacheD[y + HBS][x] = (cacheD[y + HBS][x] < cacheD[y + HBS][k] + cacheD[k][x]) ? cacheD[y + HBS][x] : cacheD[y + HBS][k] + cacheD[k][x];
		cacheD[y][x + HBS] = (cacheD[y][x + HBS] < cacheD[y][k] + cacheD[k][x + HBS]) ? cacheD[y][x + HBS] : cacheD[y][k] + cacheD[k][x + HBS];
		cacheD[y + HBS][x + HBS] = (cacheD[y + HBS][x + HBS] < cacheD[y + HBS][k] + cacheD[k][x + HBS]) ? cacheD[y + HBS][x + HBS] : cacheD[y + HBS][k] + cacheD[k][x + HBS];

		__syncthreads();
	}

	// load shared memory back to the global memory
	d[i * pitch + j] = cacheD[y][x];
	d[(i + HBS) * pitch + j] = cacheD[y + HBS][x];
	d[i * pitch + (j + HBS)] = cacheD[y][x + HBS];
	d[(i + HBS) * pitch + (j + HBS)] = cacheD[y + HBS][x + HBS];
}

__global__ void blockedFloydWarshallPhase2(int n, int blockId, int* d, size_t pitch) {
	// skipping the base block (from phase 1)
	if (blockIdx.x == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// load the base block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;

	__shared__ int cacheBaseD[BLOCK_SIZE][BLOCK_SIZE];
	cacheBaseD[y][x] = d[i * pitch + j];
	cacheBaseD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheBaseD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheBaseD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	// load the block into shared memory
	if (blockIdx.y == 0) {
		j = x + blockIdx.x * BLOCK_SIZE;
	} else {
		i = y + blockIdx.x * BLOCK_SIZE;
	}

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	cacheD[y][x] = d[i * pitch + j];
	cacheD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];
	__syncthreads();

	// compute phase 2 - partial dependent phase
	if (blockIdx.y == 0) {
		#pragma unroll 32
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			// cacheD[y][x] = min(cacheD[y][x], cacheBaseD[y][k] + cacheD[k][x]);
			cacheD[y][x] = (cacheD[y][x] < cacheBaseD[y][k] + cacheD[k][x]) ? cacheD[y][x] : cacheBaseD[y][k] + cacheD[k][x];
			cacheD[y + HBS][x] = (cacheD[y + HBS][x] < cacheBaseD[y + HBS][k] + cacheD[k][x]) ? cacheD[y + HBS][x] : cacheBaseD[y + HBS][k] + cacheD[k][x];
			cacheD[y][x + HBS] = (cacheD[y][x + HBS] < cacheBaseD[y][k] + cacheD[k][x + HBS]) ? cacheD[y][x + HBS] : cacheBaseD[y][k] + cacheD[k][x + HBS];
			cacheD[y + HBS][x + HBS] = (cacheD[y + HBS][x + HBS] < cacheBaseD[y + HBS][k] + cacheD[k][x + HBS]) ? cacheD[y + HBS][x + HBS] : cacheBaseD[y + HBS][k] + cacheD[k][x + HBS];
		}
	} else {
		#pragma unroll 32
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			// cacheD[y][x] = min(cacheD[y][x], cacheD[y][k] + cacheBaseD[k][x]);
			cacheD[y][x] = (cacheD[y][x] < cacheD[y][k] + cacheBaseD[k][x]) ? cacheD[y][x] : cacheD[y][k] + cacheBaseD[k][x];
			cacheD[y + HBS][x] = (cacheD[y + HBS][x] < cacheD[y + HBS][k] + cacheBaseD[k][x]) ? cacheD[y + HBS][x] : cacheD[y + HBS][k] + cacheBaseD[k][x];
			cacheD[y][x + HBS] = (cacheD[y][x + HBS] < cacheD[y][k] + cacheBaseD[k][x + HBS]) ? cacheD[y][x + HBS] : cacheD[y][k] + cacheBaseD[k][x + HBS];
			cacheD[y + HBS][x + HBS] = (cacheD[y + HBS][x + HBS] < cacheD[y + HBS][k] + cacheBaseD[k][x + HBS]) ? cacheD[y + HBS][x + HBS] : cacheD[y + HBS][k] + cacheBaseD[k][x + HBS];
		}
	}

	// load shared memory back to the global memory
	d[i * pitch + j] = cacheD[y][x];
	d[(i + HBS) * pitch + j] = cacheD[y + HBS][x];
	d[i * pitch + (j + HBS)] = cacheD[y][x + HBS];
	d[(i + HBS) * pitch + (j + HBS)] = cacheD[y + HBS][x + HBS];
}

__global__ void blockedFloydWarshallPhase3(int n, int blockId, int* d, size_t pitch) {
	// skipping the base blocks (from phase 1, 2)
	if (blockIdx.x == blockId || blockIdx.y == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	int i; // y + blockIdx.y * BLOCK_SIZE;
	int j; // x + blockIdx.x * BLOCK_SIZE;

	// load the base column block (same row) into shared memory
	__shared__ int cacheBaseColD[BLOCK_SIZE][BLOCK_SIZE];
	i = y + blockIdx.y * BLOCK_SIZE;
	j = x + blockId * BLOCK_SIZE;
	cacheBaseColD[y][x] = d[i * pitch + j];
	cacheBaseColD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheBaseColD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheBaseColD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	// load the base row block (same column) into shared memory
	__shared__ int cacheBaseRowD[BLOCK_SIZE][BLOCK_SIZE];
	i = y + blockId * BLOCK_SIZE;
	j = x + blockIdx.x * BLOCK_SIZE;
	cacheBaseRowD[y][x] = d[i * pitch + j];
	cacheBaseRowD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheBaseRowD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheBaseRowD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];
	__syncthreads();

	// compute phase 3 - independence phase
	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	i = y + blockIdx.y * BLOCK_SIZE;
	j = x + blockIdx.x * BLOCK_SIZE;
	cacheD[y][x] = d[i * pitch + j];
	cacheD[y + HBS][x] = d[(i + HBS) * pitch + j];
	cacheD[y][x + HBS] = d[i * pitch + (j + HBS)];
	cacheD[y + HBS][x + HBS] = d[(i + HBS) * pitch + (j + HBS)];

	#pragma unroll 32
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		// cacheD[y][x] = min(cacheD[y][x], cacheBaseColD[y][k] + cacheBaseRowD[k][x]);
		cacheD[y][x] = (cacheD[y][x] < cacheBaseColD[y][k] + cacheBaseRowD[k][x]) ? cacheD[y][x] : cacheBaseColD[y][k] + cacheBaseRowD[k][x];
		cacheD[y + HBS][x] = (cacheD[y + HBS][x] < cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x]) ? cacheD[y + HBS][x] : cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x];
		cacheD[y][x + HBS] = (cacheD[y][x + HBS] < cacheBaseColD[y][k] + cacheBaseRowD[k][x + HBS]) ? cacheD[y][x + HBS] : cacheBaseColD[y][k] + cacheBaseRowD[k][x + HBS];
		cacheD[y + HBS][x + HBS] = (cacheD[y + HBS][x + HBS] < cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x + HBS]) ? cacheD[y + HBS][x + HBS] : cacheBaseColD[y + HBS][k] + cacheBaseRowD[k][x + HBS];
	}

	// load new distance back to the global memory
	d[i * pitch + j] = cacheD[y][x];
	d[(i + HBS) * pitch + j] = cacheD[y + HBS][x];
	d[i * pitch + (j + HBS)] = cacheD[y][x + HBS];
	d[(i + HBS) * pitch + (j + HBS)] = cacheD[y + HBS][x + HBS];
}

int main(int argc, char** argv) {
	int n, m, origN;

	int* hostD;
	handleInput(argv[1], n, m, origN, &hostD);

	// pinned the host memory to accerlate cudaMemcpy
	hipHostRegister(hostD, n * n * sizeof(int), hipHostMallocDefault);

	int* deviceD;
	/* zero copy */
	// cudaHostGetDevicePointer((void**)&deviceD, (void*)hostD, cudaHostRegisterDefault);

	/* normal cuda malloc + memcpy */
	// cudaMalloc((void**)&deviceD, n * n * sizeof(int));
	// cudaMemcpy(deviceD, hostD, n * n * sizeof(int), cudaMemcpyHostToDevice);

	/* cudaMallocPitch + cudaMemcpy2D */
	size_t pitch;
	hipMallocPitch((void**)&deviceD, &pitch, n * sizeof(int), n);
	hipMemcpy2D(deviceD, pitch, hostD, n * sizeof(int), n * sizeof(int), n, hipMemcpyHostToDevice);

	/* blocked floyd warshall */

	// number of blocks is numberOfBlocks * numberOfBlocks
	// int numberOfBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	int numberOfBlocks = n / BLOCK_SIZE;

	dim3 gridPhase1(1, 1);
	dim3 gridPhase2(numberOfBlocks, 2); // the 2 represents the row & the column respectively
	dim3 gridPhase3(numberOfBlocks, numberOfBlocks);
	dim3 threadsPerBlock(32, 32);

	for (int blockId = 0; blockId < numberOfBlocks; ++blockId) {
		blockedFloydWarshallPhase1<<<gridPhase1, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase2<<<gridPhase2, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase3<<<gridPhase3, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
	}

	/* zero copy */

	/* normal cuda memcpy */
	// cudaMemcpy(hostD, deviceD, n * n * sizeof(int), cudaMemcpyDeviceToHost);

	/* cudaMemcpy2D */
	hipMemcpy2D(hostD, n * sizeof(int), deviceD, pitch, n * sizeof(int), n, hipMemcpyDeviceToHost);

	hipFree(deviceD);

	handleOutput(argv[2], n, origN, hostD);
	delete[] hostD;

	return 0;
}

// Author: justin0u0<mail@justin0u0.com>
//
// Finished basic blocked floyd warshall algorithm with no optimization


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = ((1 << 30) - 1);
const int BLOCK_SIZE = 32;

void handleInput(const char* inputFile, int& n, int& m, int** hostD) {
	FILE* file = fopen(inputFile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

	*hostD = (int*)malloc(n * n * sizeof(int));

	int** d = (int**)malloc(n * sizeof(int*));
	for (int i = 0; i < n; ++i) {
		d[i] = (*hostD) + i * n;
		for (int j = 0; j < n; ++j) {
			if (i == j) {
				d[i][j] = 0;
			} else {
				d[i][j] = INF;
			}
		}
	}

	int edge[3];
	for (int i = 0; i < m; ++i) {
		fread(edge, sizeof(int), 3, file);
		d[edge[0]][edge[1]] = edge[2];
	}
	fclose(file);
}

void handleOutput(const char* outputFile, const int n, int* hostD) {
	FILE* file = fopen(outputFile, "w");
	fwrite(hostD, sizeof(int), n * n, file);
	fclose(file);
}

__global__ void naiveFloydWarshall(const int n, const int k, int* d) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= n || y >= n) return;

	int idxIJ = x * n + y;
	int idxIK = x * n + k;
	int idxKJ = k * n + y;

	/* no cache, 6 global memory access */

	/*
		if (d[idxIJ] > d[idxIK] + d[idxKJ]) {
			d[idxIJ] = d[idxIK] + d[idxKJ];
		}
	*/

	/* shared memory cached, 2 global memory access */

	__shared__ int cacheIJ[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int cacheIK[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ int cacheKJ[BLOCK_SIZE][BLOCK_SIZE];
	cacheIJ[threadIdx.x][threadIdx.y] = d[idxIJ];
	cacheIK[threadIdx.x][threadIdx.y] = d[idxIK];
	cacheKJ[threadIdx.x][threadIdx.y] = d[idxKJ];
	__syncthreads();

	if (cacheIJ[threadIdx.x][threadIdx.y] > cacheIK[threadIdx.x][threadIdx.y] + cacheKJ[threadIdx.x][threadIdx.y]) {
		d[idxIJ] = cacheIK[threadIdx.x][threadIdx.y] + cacheKJ[threadIdx.x][threadIdx.y];
	}
}

__global__ void blockedFloydWarshallPhase1(int n, int blockId, int* d, size_t pitch) {
	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// load the block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;
	int idxIJ = i * pitch + j;

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	cacheD[y][x] = (i < n && j < n) ? d[idxIJ] : INF;
	__syncthreads();

	// compute phase 1 - dependent phase
	#pragma unroll
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		cacheD[y][x] = min(cacheD[y][x], cacheD[y][k] + cacheD[k][x]);

		__syncthreads();
	}

	// load shared memory back to the global memory
	if (i < n && j < n) {
		d[idxIJ] = cacheD[y][x];
	}
}

__global__ void blockedFloydWarshallPhase2(int n, int blockId, int* d, size_t pitch) {
	// skipping the base block (from phase 1)
	if (blockIdx.x == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// blockIdx.y: [0, 1]
	// isRow is true if the block has same index i with the base block
	bool isRow = (blockIdx.y == 0);

	// load the base block into shared memory
	int i = y + blockId * BLOCK_SIZE;
	int j = x + blockId * BLOCK_SIZE;
	int idxIJ = i * pitch + j;

	__shared__ int cacheBaseD[BLOCK_SIZE][BLOCK_SIZE];
	cacheBaseD[y][x] = (i < n && j < n) ? d[idxIJ] : INF;

	// load the block into shared memory
	if (isRow) {
		j = x + blockIdx.x * BLOCK_SIZE;
	} else {
		i = y + blockIdx.x * BLOCK_SIZE;
	}
	idxIJ = i * pitch + j;

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE];
	cacheD[y][x] = (i < n && j < n) ? d[idxIJ] : INF;
	__syncthreads();

	// compute phase 2 - partial dependent phase
	if (isRow) {
		#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			cacheD[y][x] = min(cacheD[y][x], cacheBaseD[y][k] + cacheD[k][x]);

			__syncthreads();
		}
	} else {
		#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			cacheD[y][x] = min(cacheD[y][x], cacheD[y][k] + cacheBaseD[k][x]);

			__syncthreads();
		}
	}

	// load shared memory back to the global memory
	if (i < n && j < n) {
		d[idxIJ] = cacheD[y][x];
	}
}

__global__ void blockedFloydWarshallPhase3(int n, int blockId, int* d, size_t pitch) {
	// skipping the base blocks (from phase 1, 2)
	if (blockIdx.x == blockId || blockIdx.y == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	int i = y + blockIdx.y * BLOCK_SIZE;
	int j = x + blockIdx.x * BLOCK_SIZE;
	int idxIJ = i * pitch + j;

	// load the base column block (same row) into shared memory
	__shared__ int cacheBaseColD[BLOCK_SIZE][BLOCK_SIZE];
	int baseJ = x + blockId * BLOCK_SIZE;
	cacheBaseColD[y][x] = (i < n && baseJ < n) ? d[i * pitch + baseJ] : INF;

	// load the base row block (same column) into shared memory
	__shared__ int cacheBaseRowD[BLOCK_SIZE][BLOCK_SIZE];
	int baseI = y + blockId * BLOCK_SIZE;
	cacheBaseRowD[y][x] = (baseI < n && j < n) ? d[baseI * pitch + j] : INF;
	__syncthreads();

	// compute phase 3 - independence phase

	if (i < n && j < n) {
		int curDist = d[idxIJ];

		#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			curDist = min(curDist, cacheBaseColD[y][k] + cacheBaseRowD[k][x]);
		}

		// load new distance back to the global memory
		d[idxIJ] = curDist;
	}
}

int main(int argc, char** argv) {
	int n, m;

	int* hostD;
	handleInput(argv[1], n, m, &hostD);

	// pinned the host memory to accerlate cudaMemcpy
	hipHostRegister(hostD, n * n * sizeof(int), hipHostMallocMapped);

	int* deviceD;
	/* zero copy */
	// cudaHostGetDevicePointer((void**)&deviceD, (void*)hostD, cudaHostRegisterDefault);

	/* normal cuda malloc + memcpy */
	// cudaMalloc((void**)&deviceD, n * n * sizeof(int));
	// cudaMemcpy(deviceD, hostD, n * n * sizeof(int), cudaMemcpyHostToDevice);

	/* cudaMallocPitch + cudaMemcpy2D */
	size_t pitch;
	hipMallocPitch((void**)&deviceD, &pitch, n * sizeof(int), n);
	hipMemcpy2D(deviceD, pitch, hostD, n * sizeof(int), n * sizeof(int), n, hipMemcpyHostToDevice);

	/* naive floyd warshall */

	/*
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numberOfBlocks((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
	printf("(%d, %d) (%d, %d)\n",
		threadsPerBlock.x, threadsPerBlock.y,
		numberOfBlocks.x, numberOfBlocks.y);
	for (int k = 0; k < n; ++k) {
		naiveFloydWarshall<<<numberOfBlocks, threadsPerBlock>>>(n, k, deviceD);
	}
	*/

	/* blocked floyd warshall */

	// number of blocks is numberOfBlocks * numberOfBlocks
	int numberOfBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

	dim3 gridPhase1(1, 1);
	dim3 gridPhase2(numberOfBlocks, 2); // the 2 represents the row & the column respectively
	dim3 gridPhase3(numberOfBlocks, numberOfBlocks);
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

	for (int blockId = 0; blockId < numberOfBlocks; ++blockId) {
		blockedFloydWarshallPhase1<<<gridPhase1, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase2<<<gridPhase2, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase3<<<gridPhase3, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
	}

	/* zero copy */

	/* normal cuda memcpy */
	// cudaMemcpy(hostD, deviceD, n * n * sizeof(int), cudaMemcpyDeviceToHost);

	/* cudaMemcpy2D */
	hipMemcpy2D(hostD, n * sizeof(int), deviceD, pitch, n * sizeof(int), n, hipMemcpyDeviceToHost);

	hipFree(deviceD);

	handleOutput(argv[2], n, hostD);

	return 0;
}

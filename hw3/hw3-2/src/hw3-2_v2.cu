// Author: justin0u0<mail@justin0u0.com>
//
// Finished basic blocked floyd warshall algorithm with no optimization


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = ((1 << 30) - 1);
const int BLOCK_SIZE = 32;
const int BLOCK_BASE = 5;

void handleInput(const char* inputFile, int& n, int& m, int& origN, int** hostD) {
	FILE* file = fopen(inputFile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

	origN = n;
	if (n % BLOCK_SIZE != 0) {
		n = n + (BLOCK_SIZE - n % BLOCK_SIZE);
	}

	*hostD = (int*)malloc(n * n * sizeof(int));

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			(*hostD)[i * n + j] = (i == j) ? 0 : INF;
		}
	}

	int edge[3];
	for (int i = 0; i < m; ++i) {
		fread(edge, sizeof(int), 3, file);
		(*hostD)[edge[0] * n + edge[1]] = edge[2];
	}
	fclose(file);
}

void handleOutput(const char* outputFile, const int n, const int origN, int* hostD) {
	FILE* file = fopen(outputFile, "w");
	for (int i = 0; i < origN; ++i) {
		fwrite(hostD + i * n, sizeof(int), origN, file);
	}
	// fwrite(hostD, sizeof(int), n * n, file);
	fclose(file);
}

__global__ void blockedFloydWarshallPhase1(int n, int blockId, int* d, size_t pitch) {
	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// load the block into shared memory
	// int i = y + blockId * BLOCK_SIZE;
	// int j = x + blockId * BLOCK_SIZE;
	int idxIJ = (y + (blockId << BLOCK_BASE)) * pitch + (x + (blockId << BLOCK_BASE));

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE + 1];
	cacheD[y][x] = d[idxIJ];
	__syncthreads();

	// compute phase 1 - dependent phase
	#pragma unroll
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		// cacheD[y][x] = min(cacheD[y][x], cacheD[y][k] + cacheD[k][x]);
		cacheD[y][x] = (cacheD[y][x] < cacheD[y][k] + cacheD[k][x]) ? cacheD[y][x] : cacheD[y][k] + cacheD[k][x];

		__syncthreads();
	}

	// load shared memory back to the global memory
	d[idxIJ] = cacheD[y][x];
}

__global__ void blockedFloydWarshallPhase2(int n, int blockId, int* d, size_t pitch) {
	// skipping the base block (from phase 1)
	if (blockIdx.x == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// load the base block into shared memory
	// int i = y + blockId * BLOCK_SIZE;
	// int j = x + blockId * BLOCK_SIZE;
	int idxIJ = (y + (blockId << BLOCK_BASE)) * pitch + (x + (blockId << BLOCK_BASE));

	__shared__ int cacheBaseD[BLOCK_SIZE][BLOCK_SIZE + 1];
	cacheBaseD[y][x] = d[idxIJ];

	// load the block into shared memory
	if (blockIdx.y == 0) {
		// j = x + blockIdx.x * BLOCK_SIZE;
		idxIJ = (y + (blockId << BLOCK_BASE)) * pitch + (x + (blockIdx.x << BLOCK_BASE));
	} else {
		// i = y + blockIdx.x * BLOCK_SIZE;
		idxIJ = (y + (blockIdx.x << BLOCK_BASE)) * pitch + (x + (blockId << BLOCK_BASE));
	}
	// idxIJ = i * pitch + j;

	__shared__ int cacheD[BLOCK_SIZE][BLOCK_SIZE + 1];
	cacheD[y][x] = d[idxIJ];
	__syncthreads();

	// compute phase 2 - partial dependent phase
	if (blockIdx.y == 0) {
		#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			// cacheD[y][x] = min(cacheD[y][x], cacheBaseD[y][k] + cacheD[k][x]);
			cacheD[y][x] = (cacheD[y][x] < cacheBaseD[y][k] + cacheD[k][x]) ? cacheD[y][x] : cacheBaseD[y][k] + cacheD[k][x];
		}
	} else {
		#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			// cacheD[y][x] = min(cacheD[y][x], cacheD[y][k] + cacheBaseD[k][x]);
			cacheD[y][x] = (cacheD[y][x] < cacheD[y][k] + cacheBaseD[k][x]) ? cacheD[y][x] : cacheD[y][k] + cacheBaseD[k][x];
		}
	}

	// load shared memory back to the global memory
	d[idxIJ] = cacheD[y][x];
}

__global__ void blockedFloydWarshallPhase3(int n, int blockId, int* d, size_t pitch) {
	// skipping the base blocks (from phase 1, 2)
	if (blockIdx.x == blockId || blockIdx.y == blockId) return;

	// x: [0, BLOCK_SIZE), y: [0, BLOCK_SIZE)
	int x = threadIdx.x;
	int y = threadIdx.y;

	// int i = y + blockIdx.y * BLOCK_SIZE;
	// int j = x + blockIdx.x * BLOCK_SIZE;
	int idxIJ = (y + (blockIdx.y << BLOCK_BASE)) * pitch + (x + (blockIdx.x << BLOCK_BASE));

	// load the base column block (same row) into shared memory
	__shared__ int cacheBaseColD[BLOCK_SIZE][BLOCK_SIZE + 1];
	// int baseJ = x + blockId * BLOCK_SIZE;
	cacheBaseColD[y][x] = d[(y + (blockIdx.y << BLOCK_BASE)) * pitch + (x + (blockId << BLOCK_BASE))];

	// load the base row block (same column) into shared memory
	__shared__ int cacheBaseRowD[BLOCK_SIZE][BLOCK_SIZE + 1];
	// int baseI = y + blockId * BLOCK_SIZE;
	cacheBaseRowD[y][x] = d[(y + (blockId << BLOCK_BASE)) * pitch + (x + (blockIdx.x << BLOCK_BASE))];
	__syncthreads();

	// compute phase 3 - independence phase
	int curDist = d[idxIJ];

	#pragma unroll
	for (int k = 0; k < BLOCK_SIZE; ++k) {
		// curDist = min(curDist, cacheBaseColD[y][k] + cacheBaseRowD[k][x]);
		curDist = (curDist < cacheBaseColD[y][k] + cacheBaseRowD[k][x]) ? curDist : cacheBaseColD[y][k] + cacheBaseRowD[k][x];
	}

	// load new distance back to the global memory
	d[idxIJ] = curDist;
}

int main(int argc, char** argv) {
	int n, m, origN;

	int* hostD;
	handleInput(argv[1], n, m, origN, &hostD);

	// pinned the host memory to accerlate cudaMemcpy
	hipHostRegister(hostD, n * n * sizeof(int), hipHostMallocDefault);

	int* deviceD;
	/* zero copy */
	// cudaHostGetDevicePointer((void**)&deviceD, (void*)hostD, cudaHostRegisterDefault);

	/* normal cuda malloc + memcpy */
	// cudaMalloc((void**)&deviceD, n * n * sizeof(int));
	// cudaMemcpy(deviceD, hostD, n * n * sizeof(int), cudaMemcpyHostToDevice);

	/* cudaMallocPitch + cudaMemcpy2D */
	size_t pitch;
	hipMallocPitch((void**)&deviceD, &pitch, n * sizeof(int), n);
	hipMemcpy2D(deviceD, pitch, hostD, n * sizeof(int), n * sizeof(int), n, hipMemcpyHostToDevice);

	/* blocked floyd warshall */

	// number of blocks is numberOfBlocks * numberOfBlocks
	// int numberOfBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	int numberOfBlocks = n / BLOCK_SIZE;

	dim3 gridPhase1(1, 1);
	dim3 gridPhase2(numberOfBlocks, 2); // the 2 represents the row & the column respectively
	dim3 gridPhase3(numberOfBlocks, numberOfBlocks);
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

	for (int blockId = 0; blockId < numberOfBlocks; ++blockId) {
		blockedFloydWarshallPhase1<<<gridPhase1, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase2<<<gridPhase2, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
		blockedFloydWarshallPhase3<<<gridPhase3, threadsPerBlock>>>(n, blockId, deviceD, pitch / sizeof(int));
	}

	/* zero copy */

	/* normal cuda memcpy */
	// cudaMemcpy(hostD, deviceD, n * n * sizeof(int), cudaMemcpyDeviceToHost);

	/* cudaMemcpy2D */
	hipMemcpy2D(hostD, n * sizeof(int), deviceD, pitch, n * sizeof(int), n, hipMemcpyDeviceToHost);

	hipFree(deviceD);

	handleOutput(argv[2], n, origN, hostD);

	return 0;
}
